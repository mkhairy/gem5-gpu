#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
using namespace std;
#include <hip/hip_runtime.h>
#define TIMES 24

#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////HELP FUNCTIONS/////////////////////////////////////////////////
void RandomInit(float* data, int n)
{
    for (int i=0; i<n; i++)
	{
        data[i] = rand() / (float)RAND_MAX;
	}
}

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////_VECTOR_ADDITION_///////////////////////////////////////////////////////
// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
void VectorAddition(int N, int threadsPerBlock)
{
	cout<<"Vector Addition for input size "<<N<<" :\n";
	// Variables
	float* h_A;
	float* h_B;
	float* h_C;
	float* d_A;
	float* d_B;
	float* d_C;
	double total_time=0;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Allocate vectors in device memory
    //checkCudaErrors( hipMalloc((void**)&d_A, size) );
    //checkCudaErrors( hipMalloc((void**)&d_B, size) );
    //checkCudaErrors( hipMalloc((void**)&d_C, size) );

    // Copy vectors from host memory to device memory
    //checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    //checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

	//checkCudaErrors(hipDeviceSynchronize());
    // Invoke kernel

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	for (int i = 0; i < 1; i++) {
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(h_A, h_B, h_C, N);
    getLastCudaError("kernel launch failure");
	checkCudaErrors(hipDeviceSynchronize());
	}

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

	double dSeconds = total_time/((double)TIMES * 1000);
	double dNumOps = N;
	double gflops = 1.0e-9 * dNumOps/dSeconds;
	cout<<"Time = "<<dSeconds*1.0e3<< "msec"<<endl<<"gflops = "<<gflops<<endl;

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
   // checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    
    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }

        // Free device memory
   // if (d_A)
   //     hipFree(d_A);
   // if (d_B)
  //      hipFree(d_B);
   // if (d_C)
   //     hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    //hipDeviceReset();

	if(i == N)
		cout<<"SUCCSESS"<<endl;
	else 
		cout<<"FAILED"<<endl;   
}
//////////////////////////////////////////////////////
int main(int argc,char *argv[])
{ 
  if(argc < 3)
     printf("Unsuffcient number of arguments!\n");
else
	{
		VectorAddition(atoi(argv[1]), atoi(argv[2]));
	}
}
