/*****************************************************************************
 * A microbenchmark to test the performance of varying memory copy operations
 * including different sizes and different sources and destinations
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <cassert>
#include <cmath>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#ifdef GEM5_FUSION
extern "C" {
    void m5_dumpreset_stats(uint64_t workid, uint64_t threadid);
    uint64_t rpns();
}
#endif

#define PAGE_SIZE_BYTES 4096
unsigned char touchArrayPages(unsigned char *ptr, size_t size)
{
    unsigned char sum = 0;
    for (unsigned i = 0; i < size; i += PAGE_SIZE_BYTES) {
        sum += ptr[i];
    }
    sum += ptr[size-1];
    return sum;
}

int main(int argc, char** argv) {

#ifdef GEM5GPU_AUTOMAP_COPIES
    cudaSetDeviceFlags(cudaDeviceMapHost);
#endif

    unsigned minimum_elements = 2;
    unsigned maximum_elements = 16 * 1024 * 1024;

    for (int index = 0; index < argc; ++index) {
        if (strcmp(argv[index], "-m") == 0) {
            if (argc > index+1) {
                minimum_elements = atoi(argv[++index]);
            } else {
                printf("ERROR: Must specify minimum array size to -m option\n");
                exit(0);
            }
        } else if (strcmp(argv[index], "-M") == 0) {
            if (argc > index+1) {
                maximum_elements = atoi(argv[++index]);
            } else {
                printf("ERROR: Must specify maximum array size to -M option\n");
                exit(0);
            }
        }
    }

    unsigned total_iterations = 2 * ((unsigned)log2((float)maximum_elements) - (unsigned)log2((float)minimum_elements)) + 1;
    unsigned *sizes = (unsigned*) malloc(total_iterations * sizeof(unsigned));
    double *malloc_times = (double*) malloc(total_iterations * sizeof(double));
    double *copy_times = (double*) malloc(total_iterations * sizeof(double));
    unsigned num_iterations = 0;

    bool power_two = true;
    unsigned *array = (unsigned*) malloc(maximum_elements * sizeof(unsigned));
    // In a real application, the host-side memory is likely to have been used
    // (touched) and thus, mapped before memory copies over to the device.
    unsigned hash = touchArrayPages((unsigned char*)array, maximum_elements * sizeof(unsigned));
    printf("Touch pages proof hash: %u\n", hash);
    unsigned *d_array;

    printf("Testing copy host-to-device:\n");
    printf("Size:\tMalloc (s):\tCopy (s):\tCopy (GB/s):\n");
#ifdef GEM5_FUSION
    uint64_t start;
#else
    float tmp_t;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    for (unsigned i = minimum_elements; i <= maximum_elements;) {
        sizes[num_iterations] = i;
#ifdef GEM5_FUSION
        start = rpns();
        m5_dumpreset_stats(0, 0);
#else
        hipEventRecord(start, 0);
#endif
        hipMalloc(&d_array, i * sizeof(unsigned));
#ifdef GEM5_FUSION
        m5_dumpreset_stats(0, 0);
        malloc_times[num_iterations] = (double) (rpns() - start) / 1000000000.0;
        start = rpns();
#else
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&tmp_t, start, stop);
        malloc_times[num_iterations] = (double) tmp_t / 1000.0;
        hipEventRecord(start, 0);
#endif
        hipMemcpy(d_array, array, i * sizeof(unsigned), hipMemcpyHostToDevice);
#ifdef GEM5_FUSION
        m5_dumpreset_stats(0, 0);
        copy_times[num_iterations] = (double) (rpns() - start) / 1000000000.0;
#else
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&tmp_t, start, stop);
        copy_times[num_iterations] = (double) tmp_t / 1000.0;
#endif
        if (power_two) {
            if (i > 1) {
                i = 3 * i / 2;
                power_two = false;
            } else {
                i = 2;
            }
        } else {
            i = 4 * i / 3;
            power_two = true;
        }
        num_iterations++;
    }

    // Print the copy time of the cudaMemcpyHostToDevice operations
    assert(num_iterations == total_iterations);
    for (unsigned i = 0; i < num_iterations; i++) {
        double bandwidth = ((double)(sizes[i] * 4) / copy_times[i]) / 1073741824.0;
        printf("%u\t%f\t%f\t%f\n", sizes[i], malloc_times[i], copy_times[i], bandwidth);
    }

    num_iterations = 0;
    power_two = true;

    printf("\n\nTesting copy device-to-host:\n");
    printf("Size:\tMalloc (s):\tCopy (s):\tCopy (GB/s):\n");
    for (unsigned i = minimum_elements; i <= maximum_elements;) {
        sizes[num_iterations] = i;
#ifdef GEM5_FUSION
        start = rpns();
        m5_dumpreset_stats(0, 0);
#else
        hipEventRecord(start, 0);
#endif
        hipMalloc(&d_array, i * sizeof(unsigned));
#ifdef GEM5_FUSION
        m5_dumpreset_stats(0, 0);
        malloc_times[num_iterations] = (double) (rpns() - start) / 1000000000.0;
        start = rpns();
#else
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&tmp_t, start, stop);
        malloc_times[num_iterations] = (double) tmp_t / 1000.0;
        hipEventRecord(start, 0);
#endif
        hipMemcpy(array, d_array, i * sizeof(unsigned), hipMemcpyDeviceToHost);
#ifdef GEM5_FUSION
        m5_dumpreset_stats(0, 0);
        copy_times[num_iterations] = (double) (rpns() - start) / 1000000000.0;
#else
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&tmp_t, start, stop);
        copy_times[num_iterations] = (double) tmp_t / 1000.0;
#endif
        if (power_two) {
            if (i > 1) {
                i = 3 * i / 2;
                power_two = false;
            } else {
                i = 2;
            }
        } else {
            i = 4 * i / 3;
            power_two = true;
        }
        num_iterations++;
    }

    // Print the copy time of the cudaMemcpyDeviceToHost operations
    assert(num_iterations == total_iterations);
    for (unsigned i = 0; i < num_iterations; i++) {
        double bandwidth = ((double)(sizes[i] * 4) / copy_times[i]) / 1073741824.0;
        printf("%u\t%f\t%f\t%f\n", sizes[i], malloc_times[i], copy_times[i], bandwidth);
    }
    printf("\n");

    free(array);
    free(malloc_times);
    free(copy_times);
    hipFree(d_array);

    return 0;
}
