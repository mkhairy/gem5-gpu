
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define MAGIC 1234

__global__ void kernel(int *array) {
	int value = *array;
        value += 1;
	*array = value;
}



int main() {

	int global = MAGIC;
	int *d_global;
	hipMalloc(&d_global, sizeof(int));
	hipMemcpy(d_global, &global, sizeof(int), hipMemcpyHostToDevice);

	kernel<<<1,1>>>(d_global);

	hipMemcpy(&global, d_global, sizeof(int), hipMemcpyDeviceToHost);

	if (global == MAGIC+1) {
		cout << "Test Passed" << endl;
	} else {
		cout << "Test Failed " << global << endl;
	}
	return 0;
}
