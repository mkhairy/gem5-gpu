
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define MAGIC 1234

__global__ void kernel(int *array) {
	int value = *array;
        value += 1;
	*array = value;
}

int main() {

	int *global;
	int *d_global;

	hipHostMalloc((void**)&global, sizeof(int), hipHostMallocDefault);
	*global = MAGIC;
	hipMalloc(&d_global, sizeof(int));

	hipMemcpy(d_global, global, sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1,1>>>(d_global);
	hipMemcpy(global, d_global, sizeof(int), hipMemcpyDeviceToHost);

	if (*global == MAGIC+1) {
		cout << "Test Passed" << endl;
	} else {
		cout << "Test Failed " << global << endl;
	}

    hipHostFree(global);
	hipFree(d_global);

	return 0;
}
