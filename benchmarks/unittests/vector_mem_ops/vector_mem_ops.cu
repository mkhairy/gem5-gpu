

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void kernel4(int4 *input4, int4 *output4) {
	*output4 = *input4;
}

__global__ void kernel2(int2 *input2, int2 *output2) {
	*output2 = *input2;
}

__global__ void kernelArr4(int4 *input4, int4 *output4) {
	int tid = threadIdx.x;
	output4[tid] = input4[tid];
}

int main() {
	int4 *input4, *output4;
	int4 *dinput4, *doutput4;

	input4 = (int4*)malloc(sizeof(int4));
	output4 = (int4*)malloc(sizeof(int4));

	hipMalloc(&dinput4, sizeof(int4));
	hipMalloc(&doutput4, sizeof(int4));

	input4->x = 1;
	input4->y = 2;
	input4->z = 3;
	input4->w = 4;
	output4->x = 42;
	output4->y = 43;
	output4->z = 44;
	output4->w = 45;

	hipMemcpy(dinput4, input4, sizeof(int4), hipMemcpyHostToDevice);
	hipMemcpy(doutput4, output4, sizeof(int4), hipMemcpyHostToDevice);

	kernel4<<<1,1>>>(dinput4, doutput4);

	hipMemcpy(output4, doutput4, sizeof(int4), hipMemcpyDeviceToHost);

	bool passed = true;
	if (input4->x != output4->x) {
		cerr << "output4->x wrong! " << output4->x << endl;
		passed = false;
	}
	if (input4->y != output4->y) {
		cerr << "output4->y wrong! " << output4->y << endl;
		passed = false;
	}
	if (input4->z != output4->z) {
		cerr << "output4->z wrong! " << output4->z << endl;
		passed = false;
	}
	if (input4->w != output4->w) {
		cerr << "output4->w wrong! " << output4->w << endl;
		passed = false;
	}

	if (passed) {
		cout << "Vector 4 Test passed" << endl;
	}

	int2 *input2, *output2;
	int2 *dinput2, *doutput2;

	input2 = (int2*)malloc(sizeof(int2));
	output2 = (int2*)malloc(sizeof(int2));

	hipMalloc(&dinput2, sizeof(int2));
	hipMalloc(&doutput2, sizeof(int2));

	input2->x = 1;
	input2->y = 2;
	output2->x = 42;
	output2->y = 43;

	hipMemcpy(dinput2, input2, sizeof(int2), hipMemcpyHostToDevice);
	hipMemcpy(doutput2, output2, sizeof(int2), hipMemcpyHostToDevice);

	kernel2<<<1,1>>>(dinput2, doutput2);

	hipMemcpy(output2, doutput2, sizeof(int2), hipMemcpyDeviceToHost);

	bool passed2 = true;
	if (input2->x != output2->x) {
		cerr << "output2->x wrong! " << output2->x << endl;
		passed2 = false;
	}
	if (input2->y != output2->y) {
		cerr << "output2->y wrong! " << output2->y << endl;
		passed2 = false;
	}

	if (passed2) {
		cout << "Vector 2 Test passed" << endl;
	} else {
		passed = false;
	}

	int4 *inputArr4, *outputArr4;
	int4 *dinputArr4, *doutputArr4;

	inputArr4 = (int4*)malloc(sizeof(int4)*32);
	outputArr4 = (int4*)malloc(sizeof(int4)*32);

	hipMalloc(&dinputArr4, sizeof(int4)*32);
	hipMalloc(&doutputArr4, sizeof(int4)*32);

	for (int i=0; i<32; i++) {
		inputArr4[i].x = 5;
		inputArr4[i].y = 2+i*4;
		inputArr4[i].z = 3+i*4;
		inputArr4[i].w = 4+i*4;
		outputArr4[i].x = 4001+i*4;
		outputArr4[i].y = 4002+i*4;
		outputArr4[i].z = 4003+i*4;
		outputArr4[i].w = 4004+i*4;
	}

	hipMemcpy(dinputArr4, inputArr4, sizeof(int4)*32, hipMemcpyHostToDevice);
	hipMemcpy(doutputArr4, outputArr4, sizeof(int4)*32, hipMemcpyHostToDevice);

	kernelArr4<<<1,32>>>(dinputArr4, doutputArr4);

	hipMemcpy(outputArr4, doutputArr4, sizeof(int4)*32, hipMemcpyDeviceToHost);

	bool passedArr = true;
	for (int i=0; i<32; i++) {
		if (outputArr4[i].x != inputArr4[i].x) {
			cerr << "outputArr4[" << i << "].x wrong! " << outputArr4[i].x << endl;
			passedArr = false;
		}
		if (outputArr4[i].y != inputArr4[i].y) {
			cerr << "outputArr4[" << i << "].y wrong! " << outputArr4[i].y << endl;
			passedArr = false;
		}
		if (outputArr4[i].z != inputArr4[i].z) {
			cerr << "outputArr4[" << i << "].z wrong! " << outputArr4[i].z << endl;
			passedArr = false;
		}
		if (outputArr4[i].w != inputArr4[i].w) {
			cerr << "outputArr4[" << i << "].w wrong! " << outputArr4[i].w << endl;
			passedArr = false;
		}
	}

	if (passedArr) {
		cout << "Vector Array Test passed" << endl;
	} else {
		passed = false;
	}

	if (passed)
		return 0;
	else
		return 1;
}