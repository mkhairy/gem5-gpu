


#include <hip/hip_runtime.h>
#include <iostream>

#define MAGIC 1234

using namespace std;

__constant__ int constantMem[1];


__global__ void kernel(int *out) {

	*out = constantMem[0];

}



int main() {

	int* input = (int*)malloc(sizeof(int));

	*input = MAGIC;

	hipMemcpyToSymbol(HIP_SYMBOL(constantMem), input, sizeof(int));

	int *dout;

	hipMalloc(&dout, sizeof(int));

	hipMemset(dout, 0, sizeof(int));

	kernel<<<1,1>>>(dout);

	int *out = (int*)malloc(sizeof(int));

	hipMemcpy(out, dout, sizeof(int), hipMemcpyDeviceToHost);

	if (*out == MAGIC) {
		cout << "Test Passed" << endl;
	} else {
		cout << "Test Failed " << *out << endl;
	}


	return 0;
}