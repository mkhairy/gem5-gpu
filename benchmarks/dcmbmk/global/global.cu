#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdint.h>

#include "../common/repeat.h"

const int page_size = 4;	// Scale stride and arrays by page size.


__global__ void global_latency (unsigned int ** my_array, int array_length, int iterations, int ignore_iterations, unsigned long long * duration) {

	unsigned int start_time, end_time;
	unsigned int *j = (unsigned int*)my_array; 
	volatile unsigned long long sum_time;

	sum_time = 0;
	duration[0] = 0;

	for (int k = -ignore_iterations; k < iterations; k++) {
		if (k==0) {
			sum_time = 0; // ignore some iterations: cold icache misses
		}

		start_time = clock();
		repeat256(j=*(unsigned int **)j;)
		end_time = clock();

		sum_time += (end_time - start_time);
	}

	((unsigned int*)my_array)[array_length] = (unsigned int)j;
	((unsigned int*)my_array)[array_length+1] = (unsigned int) sum_time;
	duration[0] = sum_time;
}

int gcf(int a, int b)
{
	if (a == 0) return b;
	return gcf(b % a, a);
}

/* Construct an array of N unsigned ints, with array elements initialized
   so kernel will make stride accesses to the array. Then launch kernel
   10 times, each making iterations*256 global memory accesses. */
void parametric_measure_global(int N, int iterations, int ignore_iterations, int stride) {
	
	int i;
	unsigned int * h_a;
	unsigned int ** d_a;

	unsigned long long * duration;
	unsigned long long * latency;
	unsigned long long latency_sum = 0;

	// Don't die if too much memory was requested.
	if (N > 241600000) { printf ("OOM.\n"); return; }

	/* allocate arrays on CPU */
	h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N+2));
	latency = (unsigned long long *)malloc(sizeof(unsigned long long));

	/* allocate arrays on GPU */
	hipMalloc ((void **) &d_a, sizeof(unsigned int) * (N+2));
	hipMalloc ((void **) &duration, sizeof(unsigned long long));

   	/* initialize array elements on CPU with pointers into d_a. */
	
	int step = gcf (stride, N);	// Optimization: Initialize fewer elements.
	for (i = 0; i < N; i += step) {
		// Device pointers are 32-bit on GT200.
		h_a[i] = ((unsigned int)(uintptr_t)d_a) + ((i + stride) % N)*sizeof(unsigned int);	
	}

	h_a[N] = 0;
	h_a[N+1] = 0;


	hipDeviceSynchronize ();

        /* copy array elements from CPU to GPU */
        hipMemcpy((void *)d_a, (void *)h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice);

	hipDeviceSynchronize ();


	/* Launch a multiple of 10 iterations of the same kernel and take the average to eliminate interconnect (TPCs) effects */

	for (int l=0; l <10; l++) {
	
		/* launch kernel*/
		dim3 Db = dim3(1);
		dim3 Dg = dim3(1,1,1);

		//printf("Launch kernel with parameters: %d, N: %d, stride: %d\n", iterations, N, stride); 
		global_latency <<<Dg, Db>>>(d_a, N, iterations, ignore_iterations, duration);

		hipDeviceSynchronize ();

		hipError_t error_id = hipGetLastError();
        	if (error_id != hipSuccess) {
			printf("Error is %s\n", hipGetErrorString(error_id));
		}

		/* copy results from GPU to CPU */
		hipDeviceSynchronize ();

	        //hipMemcpy((void *)h_a, (void *)d_a, sizeof(unsigned int) * (N+2), hipMemcpyDeviceToHost);
        	hipMemcpy((void *)latency, (void *)duration, sizeof(unsigned long long), hipMemcpyDeviceToHost);

	        hipDeviceSynchronize ();
		latency_sum+=latency[0];

	}

	/* free memory on GPU */
	hipFree(d_a);
	hipFree(duration);
	hipDeviceSynchronize ();

        /*free memory on CPU */
        free(h_a);
        free(latency);

	printf("%f\n", (double)(latency_sum/(10*256.0*iterations)) );

}



/* Test page size. Construct an access pattern of N elements spaced stride apart,
   followed by a gap of stride+offset, followed by N more elements spaced stride
   apart. */
void measure_pagesize(int N, int stride, int offset) {
	
	unsigned int ** h_a;
	unsigned int ** d_a;

	unsigned long long * duration;
	unsigned long long * latency;

	unsigned long long latency_sum = 0;
	
	const int size = N * stride * 2 + offset + stride*2;
	const int iterations = 20;

	// Don't die if too much memory was requested.
	if (size > 241600000) { printf ("OOM.\n"); return; }

	/* allocate array on CPU */
	h_a = (unsigned int **)malloc(4 * size);
	latency = (unsigned long long *)malloc(sizeof(unsigned long long));

	/* allocate array on GPU */
	hipMalloc ((void **) &d_a, sizeof(unsigned int) * size);
	hipMalloc ((void **) &duration, sizeof(unsigned long long));

   	/* initialize array elements on CPU */

	for (int i=0;i<N; i++)
		((unsigned int *)h_a)[i*stride] = ((i*stride + stride)*4) + (uintptr_t) d_a;

	((unsigned int *)h_a)[(N-1)*stride] = ((N*stride + offset)*4) + (uintptr_t) d_a;	//point last element to stride+offset

	for (int i=0;i<N; i++)
		((unsigned int *)h_a)[(i+N)*stride+offset] = (((i+N)*stride + offset + stride)*4) + (uintptr_t) d_a;

	((unsigned int *)h_a)[(2*N-1)*stride+offset] = (uintptr_t) d_a;		//wrap around.
	


        hipDeviceSynchronize ();

        /* copy array elements from CPU to GPU */
        hipMemcpy((void *)d_a, (void *)h_a, sizeof(unsigned int) * size, hipMemcpyHostToDevice);
        
	hipDeviceSynchronize ();


	for (int l=0; l < 10 ; l++) {
	
		/* launch kernel*/
		dim3 Db = dim3(1);
		dim3 Dg = dim3(1,1,1);

		//printf("Launch kernel with parameters: %d, N: %d, stride: %d\n", iterations, N, stride); 
		global_latency <<<Dg, Db>>>(d_a, N, iterations, 1, duration);

		hipDeviceSynchronize ();

		hipError_t error_id = hipGetLastError();
	        if (error_id != hipSuccess) {
			printf("Error is %s\n", hipGetErrorString(error_id));
		}

		/* copy results from GPU to CPU */
		hipDeviceSynchronize ();

	        //hipMemcpy((void *)h_a, (void *)d_a, sizeof(unsigned int) * N, hipMemcpyDeviceToHost);
	        hipMemcpy((void *)latency, (void *)duration, sizeof(unsigned long long), hipMemcpyDeviceToHost);

        	hipDeviceSynchronize ();

		latency_sum+=latency[0];
	}

	/* free memory on GPU */
	hipFree(d_a);
	hipFree(duration);
	hipDeviceSynchronize ();


        /*free memory on CPU */
        free(h_a);
        free(latency);
	

	printf("%f\n", (double)(latency_sum/(10.0*256*iterations)));
}




void measure_global1() {

	// we will measure latency of global memory
	// One thread that accesses an array.
	// loads are dependent on the previously loaded values

	int N, iterations, stride; 

	// initialize upper bounds here
	int stride_upper_bound; 

	printf("Global1: Global memory latency for 1 KB array and varying strides.\n");
	printf("   stride (bytes), latency (clocks)\n");


	N=256;		// 131072;
	iterations = 4;
	stride_upper_bound = N; 
	for (stride = 1; stride <= (stride_upper_bound) ; stride+=1) {
		printf ("  %5d, ", stride*4);
		parametric_measure_global(N, iterations, 1, stride);
	}
}


void measure_global5() {

	int N, iterations, stride; 

	// initialize upper bounds here

	printf("\nGlobal5: Global memory latency for %d KB stride.\n", 512 * page_size/4);
	printf("   Array size (KB), latency (clocks)\n");


	iterations = 1;
	stride = 512 * 1024 / 4;
	for (N = (1*1024*1024); N <= (16*1024*1024); N += stride) {
		printf ("   %5d, ", N*4/1024 * page_size/4);
		parametric_measure_global(N*page_size/4, iterations, 1, stride *page_size/4);
	}
}

void measure_global6() {
	int N, stride, entries;
	
	printf("\nGlobal6: Testing associativity of L1 TLB.\n");
	printf("   entries, array size (KB), stride (KB), latency\n");

	for (entries = 16; entries <= 17; entries++) {
		for (stride = 1; stride <= (4*1024*1024); stride *= 2 ) {
			for (int substride = 1; substride < 16; substride *= 2 ) {
				int stride2 = stride * sqrt(sqrt(substride)) + 0.5;
				N = entries * stride2;
				
				printf ("   %d, %7.2f, %7f, ", entries, N*4/1024.0*page_size/4, stride2*4/1024.0*page_size/4);
				parametric_measure_global(N*page_size/4, 4, 1, stride2*page_size/4);
			}
		}
	}
}

void measure_global4()
{
	printf ("\nGlobal4: Measuring L2 TLB page size using %d MB stride\n", 2 * page_size/4);
	printf ("  offset (bytes), latency (clocks)\n");
		
	// Small offsets (approx. page size) are interesting. Search much bigger offsets to
	// ensure nothing else interesting happens.
	for (int offset = -2048/4; offset <= (2097152+1536)/4; offset += (offset < 1536) ? 128/4 : 4096/4)
	{
		printf ("  %d, ", offset*4 *page_size/4);
		measure_pagesize(10, 2097152/4 *page_size/4, offset* page_size/4);
	}
	
}

int main() {
	printf("Assuming page size is %d KB\n", page_size);
	measure_global1();
	measure_global4();
	measure_global5();
	measure_global6();
	return 0;
}
